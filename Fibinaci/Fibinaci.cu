#include "hip/hip_runtime.h"
#include "Fibinaci.h"

#include "hip/hip_runtime.h"
#include ""



__global__ void CalculateFibi(int n,int *seq)
{
	int t1 = 0, t2 = 1, nextTerm = 0;
	int i = threadIdx.x;

	for (int q = 0; q < i; q++) 
	{
		nextTerm = t1 + t2;
		t1 = t2;
		t2 = nextTerm;

	}
	
	
	
}

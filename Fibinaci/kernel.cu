﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include<iostream>
__global__ void CalculateFibi(long long n, long long *seq)
{
	long long t1 = 0, t2 = 1, nextTerm = 0;
	int i = threadIdx.x;

		for (long q = 0; q < i; q++)
		{

			nextTerm = t1 + t2;
			t1 = t2;
			t2 = nextTerm;

		}
		seq[i] = nextTerm;
	



}

int main()
{
	long long* seq;

	

	int n;
	std::cout << "Enter the number of terms: ";
	std::cin >> n;


	hipMallocManaged(&seq, n*sizeof(long long));
	for (long i = 0; i < n; i++) {
		seq[i] = 0;
	}

	CalculateFibi<<<1,256>>>(n,seq);
	hipDeviceSynchronize();
	

	
	std::cout << "Fibonacci Series: ";
	for (int i = 0; i < n; i++)
		printf("Seq[%d] = %d\n", i, seq[i]);


	hipFree(seq);

	
	
 
}

